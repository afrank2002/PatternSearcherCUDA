#include "hip/hip_runtime.h"
﻿#include ""
#include <hip/hip_runtime.h>

#include <stdio.h>

#include <hip/hip_runtime.h>
#include <iostream>
#include <string>
#include <fstream>
#include <cstring>

using namespace std;

// Kernel to search for patterns in rows of data
// Kernel to search for patterns in rows of data
__global__ void searchPattern(const char* inputRows, int numRows, int numColumns, const char* pattern, int patternLength, int* results) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    printf("TID: %d\n", tid);
    if (tid < numRows) {
        // Calculate the starting index of the row
        int rowStartIndex = tid * numColumns;

        // Get a pointer to the start of the row
        const char* row = inputRows + rowStartIndex;

        int matchCount = 0;
        printf("Thread %d is processing row %d\n", tid, tid + 1);
        printf("Row content: ");
        for (int k = 0; k < numColumns; ++k) {
            printf("%c", row[k]);
        }
        printf("\n");
        for (int i = 0; i <= numColumns - patternLength; ++i) {
            int columnNumber;
          //  printf("Thread %d is checking position %d in row %d\n", tid, i, tid + 1);
            bool match = true;
         //   printf("pattern length is %d\n", patternLength);
            for (int j = 0; j < patternLength; ++j) {
                // printf("TID: %d  COMPARING %d TO %d\n", tid, row[i + j], pattern[j]);
                if (row[i + j] != pattern[j]) {
                  //  printf("TID: %d  PATTERN BROKEN AT %d , %d\n", tid, tid + 1, i + j + tid + 1);
                    match = false;
                    break;
                }
                else if (row[i + j] == pattern[j] && j == 0)
                {
                    columnNumber = i + j + 1;
                    //printf("TID: %d  PATTERN FOUND AT %d , %d\n", tid, tid + 1, i + j + 1);
                }
                else if (row[i + j] == pattern[j] && j == patternLength - 1)
                {
                  // printf("TID: %d  COMPLETED PATTERN AT %d , %d\n", tid, tid + 1, i + j + 1);
                }
            }
            if (match) {
                // Calculate the position of the match within the row
                // Store the position of the match in the results array along with the row and column numbers
                results[(tid * numColumns + matchCount) * 2] = tid + 1; // Row number
                printf("%d\n", (tid * numColumns + matchCount) * 2);

                results[(tid * numColumns + matchCount) * 2 + 1] = columnNumber; // Column number
                printf("%d\n", (tid * numColumns + matchCount) * 2 + 1);
                matchCount++;
                printf("Thread %d found a match at position (%d, %d)\n", tid, tid + 1, columnNumber);
            }
        }
        // Mark the end of matches for this row
        results[(tid * numColumns + matchCount) * 2] = -1;
        results[(tid * numColumns + matchCount) * 2 + 1] = -1;
    }
}




int main() {
    // Define input, pattern, and output file paths
    string inputFileName = "C:/Users/MaristUser/source/repos/PatternSearcher/x64/Debug/input.txt";
    string patternFileName = "C:/Users/MaristUser/source/repos/PatternSearcher/x64/Debug/pattern.txt";
    string outputFileName = "C:/Users/MaristUser/source/repos/PatternSearcher/x64/Debug/output.txt";

    // Open input, pattern, and output files
    ifstream inputFile(inputFileName);
    ifstream patternFile(patternFileName);
    ofstream outputFile(outputFileName);

    // Check if files are opened successfully
    if (!inputFile || !patternFile || !outputFile) {
        cerr << "Error opening files." << endl;
        return 1;
    }

    // Read the first line of the input and pattern files
    string inputLine, patternLine;
    getline(patternFile, patternLine);
    // Calculate the number of columns and rows in the input file
    int numColumns;
    int numRows = 0;

    string inputString;
    while (getline(inputFile, inputLine)) {
        inputString += inputLine; // Append each line to the inputString with a newline character
        ++numRows;
    }
    numColumns = inputLine.length();

    printf("the number of rows is %d\n", numRows);

    // Allocate memory on GPU
    char* inputRowsDevice;
    char* patternDevice;
    int* resultsDevice;
    int* resultsHost = new int[numRows * numColumns * 2]; // Results array stores both row and column numbers

    hipMalloc((void**)&inputRowsDevice, numRows * numColumns * sizeof(char));
    hipMalloc((void**)&patternDevice, patternLine.length() * sizeof(char));
    hipMalloc((void**)&resultsDevice, numRows * numColumns * 2 * sizeof(int)); // Each match has two integers (row and column)

    // Copy input data and pattern from CPU to GPU
    hipMemcpy(inputRowsDevice, inputString.c_str(), numRows * numColumns * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(patternDevice, patternLine.c_str(), patternLine.length() * sizeof(char), hipMemcpyHostToDevice);

    // Invoke the CUDA kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (numRows + threadsPerBlock) / threadsPerBlock;
    printf("row: %s\n", inputString.c_str());
    searchPattern << <1, numRows >> > (inputRowsDevice, numRows, numColumns, patternDevice, patternLine.length(), resultsDevice);

    // Copy results back from GPU to CPU
    hipMemcpy(resultsHost, resultsDevice, numRows * numColumns * 2 * sizeof(int), hipMemcpyDeviceToHost);
 

    // Process results on CPU and write to output file
    for (int i = 0; i < numRows * numColumns * 2; i += 2) {
        int row = resultsHost[i];
        printf("ROW: %d ", row);

        int col = resultsHost[i + 1];
        printf(" COL: %d ", col);

        if (row > 0 && col > 0) {
            outputFile << "Pattern found at position: (" << row << ", " << col << ")" << endl;
        }
    }

    // Free allocated memory on GPU
    hipFree(inputRowsDevice);
    hipFree(patternDevice);
    hipFree(resultsDevice);

    // Close files
    inputFile.close();
    patternFile.close();
    outputFile.close();

    // Deallocate memory
    delete[] resultsHost;

    return 0;
}

